#include "hip/hip_runtime.h"
// game_of_life.c
//
// Nvidia CUDA implementation of simultaneous game of life.
//
// Created by Filip Jany & Patryk Stopyra
// Wroclaw, 22.05.2015.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"

#define PARAMS_NUMBER 6

typedef struct {
    int width;
    int height;
    int startTry;
    int endTry;
    char *path;
	int pathlen;
	char *src;
} params;

params parseParams(int argc, char **argv) {
    params p;

    sscanf(argv[1], "%d", &(p.width));

    sscanf(argv[2], "%d", &(p.height));
    sscanf(argv[3], "%d", &(p.startTry));
    sscanf(argv[4], "%d", &(p.endTry));

    p.path = (char*) malloc((strlen(argv[5])+16) * sizeof(char));
    strncpy(p.path, argv[5], strlen(argv[5]));

	p.pathlen = strlen(p.path);

	if(argc > 6)
	{
		p.src = (char*) malloc((strlen(argv[6])+1) * sizeof(char));
		strncpy(p.src, argv[6], strlen(argv[6]));
	}	
	else
		p.src = NULL;
    return p;
}

int validateParams(params p) {
    if (p.width % BOARD_TYPE_LENGTH != 0) {
        printf("[ERROR] Width has to be a multiplication of %d.\n", BOARD_TYPE_LENGTH);
        return 0;
    }

    if (p.height % BOARD_TYPE_LENGTH != 0) {
        printf("[ERROR] Height has to be a multiplication of %d.\n", BOARD_TYPE_LENGTH);
        return 0;
    }

    return 1;
}

void userInfo() {
    printf("Proper usage:\n\tgame_of_life [width] [height] [start] [end] [path]\n");
    printf("where:\n");
    printf("\twidth - width of torus field\n");
    printf("\theight - height of torus field\n");
    printf("\tstart - round of recording begin\n");
    printf("\tend - round of recording stop\n");
    printf("\tpath - destination of recorded rounds\n");
}

int main(int argc, char **argv) {
    if (argc < PARAMS_NUMBER) {
        userInfo();
        return 1;
    }

    params p = parseParams(argc, argv);
    if (!validateParams(p))
        return 2;
	universe *uni;// = prepareUniverse(p.width, p.height);
	if(argc == 7)
		uni = prepareUniverseFromSource(p.width, p.height, p.src);
	else
		uni = prepareUniverse(p.width, p.height);

	world* w = copyArrayToDevice(*uni);
	
	for(int i = 0; i <= p.endTry; ++i)
	{
		if(i >= p.startTry)
		{
			copyArrayToHost(w, uni);
			//char* currentName = malloc(sizeof(char)
			//sprintf()
			sprintf(p.path + p.pathlen, "_%d.txt", i);
			saveToFile(uni, p.path);
		}
		computeNextStep<<<32,1>>>(w);
		w->actual = !w->actual;
	}
    //printf("--- %d\n", -5 % 3);
    //printf("%d %d %d %d %s\n", p.width, p.height, p.startTry, p.endTry, p.path);
	destroyUniverse(uni);
	free(w);
    return 0;
}