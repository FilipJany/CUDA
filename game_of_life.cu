#include "hip/hip_runtime.h"
// game_of_life.c
//
// Nvidia CUDA implementation of simultaneous game of life.
//
// Created by Filip Jany & Patryk Stopyra
// Wroclaw, 22.05.2015.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"
#include <time.h>

#define PARAMS_NUMBER 6

typedef struct {
    int width;
    int height;
    int startTry;
    int endTry;
    char *path;
	int pathlen;
	char *src;
} params;

params parseParams(int argc, char **argv) {
    params p;

    sscanf(argv[1], "%d", &(p.width));

    sscanf(argv[2], "%d", &(p.height));
    sscanf(argv[3], "%d", &(p.startTry));
    sscanf(argv[4], "%d", &(p.endTry));

    p.path = (char*) malloc((strlen(argv[5])+16) * sizeof(char));
    strncpy(p.path, argv[5], strlen(argv[5]));

	p.pathlen = strlen(p.path);

	if(argc > 6)
	{
		p.src = (char*) malloc((strlen(argv[6])+1) * sizeof(char));
		strncpy(p.src, argv[6], strlen(argv[6]));
	}	
	else
		p.src = NULL;
    return p;
}

int validateParams(params p) {
    if (p.width % BOARD_TYPE_LENGTH != 0) {
        printf("[ERROR] Width has to be a multiplication of %d.\n", BOARD_TYPE_LENGTH);
        return 0;
    }

    if (p.height % BOARD_TYPE_LENGTH != 0) {
        printf("[ERROR] Height has to be a multiplication of %d.\n", BOARD_TYPE_LENGTH);
        return 0;
    }

    return 1;
}

void userInfo() {
    printf("Proper usage:\n\tgame_of_life [width] [height] [start] [end] [path]\n");
    printf("where:\n");
    printf("\twidth - width of torus field\n");
    printf("\theight - height of torus field\n");
    printf("\tstart - round of recording begin\n");
    printf("\tend - round of recording stop\n");
    printf("\tpath - destination of recorded rounds\n");
}

int main(int argc, char **argv) {
    if (argc < PARAMS_NUMBER) {
        userInfo();
        return 1;
    }

    params p = parseParams(argc, argv);
    if (!validateParams(p))
        return 2;
	universe *uni;// = prepareUniverse(p.width, p.height);
	if(argc == 7)
		uni = prepareUniverseFromSource(p.width, p.height, p.src);
	else
		uni = prepareUniverse(p.width, p.height);
	
	int* tab_0;
	int* tab_1;
	hipMalloc(&(tab_0), uni->height*uni->width / sizeof(int));
	hipMalloc(&(tab_1), uni->height*uni->width / sizeof(int));
	copyArrayToDevice(*uni, tab_0);
	int actual = 0;
	clock_t begin, end;
	double time_spent;

	begin = clock();

	for(int i = 0; i <= p.endTry; ++i)
	{
		if(i >= p.startTry)
		{
			copyArrayToHost(uni, tab_0, tab_1, actual);
			sprintf(p.path + p.pathlen, "_%d.txt", i);
			saveToFile(uni, p.path);
		}
		//computeNextStepSharedMemory<<<512, 32, uni->width * (uni->height / BOARD_TYPE_LENGTH)>>>(tab_0, tab_1, actual);
		computeNextStep<<<32,1>>>(tab_0, tab_1, actual);
		hipDeviceSynchronize();
		actual = !actual;
	}
	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Time spent: %lf.\n", time_spent);
    destroyUniverse(uni);
    return 0;
}