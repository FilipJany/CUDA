#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"


static int width;
static int height;


//------ Some computational functions ------//

int mod(int number, int modulus) {
	return abs(number % modulus);
}

void computeNextStep(int* board_d)
{
	//do the magic
}

__global__ int getMooreNeighborhood(int* board_d)
{
    int numNbrs = 0;

    int x = (numer porzadkowy wierzcholka) / (height * BOARD_TYPE_LENGTH);
    int y = ((numer porzadkowy wierzcholka) - x * (height * BOARD_TYPE_LENGTH));

    return numNbrs;
}

int* copyArrayToDevice(universe uni)
{
	int width = uni.width; //number of int's in array -> width
	int* board_d;
	hipMalloc((void**)&board_d, uni.height*uni.width/sizeof(int));
	
	for(int i=0; i < uni.width; ++i)
	{
		hipMemcpy(board_d+i*uni.height/BOARD_TYPE_LENGTH, uni.board[i], uni.height/BOARD_TYPE_LENGTH, hipMemcpyHostToDevice);
	}
	width = uni.width;
	height = uni.height;
	return board_d;
}

void copyArrayToHost(int* board_d, universe uni)
{
	for(int i = 0; i < uni.width; ++i)
	{
		hipMemcpy(uni.board[i], board_d+i*uni.height/BOARD_TYPE_LENGTH, uni.height/BOARD_TYPE_LENGTH, hipMemcpyDeviceToHost);
	}
}
