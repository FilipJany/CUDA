#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"


__device__ static int width;
__device__ static int height;
__device__ static int size;
extern __shared__ int sharedArray[];


//------ Some computational functions ------//

__device__ int mod(int number, int modulus) {
	if (number % modulus >= 0)
		return number % modulus;
	else
		return modulus + (number % modulus);
}

__device__ int computeMidCells(int left, int mid, int right) {
	int result = 0;

	for (int i = 1; i < BOARD_TYPE_LENGTH - 1; ++i) {
		//printf("left: %d, mid: %d, right: %d\n", left, mid, right);
		int j = i - 1;
		int k = i + 1;

		int sum = (left & (1 << j) ? 1 : 0)
			+ (left & (1 << i) ? 1 : 0)
			+ (left & (1 << k) ? 1 : 0)
			+ (mid & (1 << j) ? 1 : 0)
			+ (mid & (1 << k) ? 1 : 0)
			+ (right & (1 << j) ? 1 : 0)
			+ (right & (1 << i) ? 1 : 0)
			+ (right & (1 << k) ? 1 : 0);
		//if (blockIdx.x > 9 && blockIdx.x < 14)
			//printf("%d\t", sum);
		if (mid & (1 << i)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << i);
			}
		}
		else {
			
			if (sum == 3) {
				//printf("aliveCMC: %d\n", mid);
				result = result | (1 << i);
			}
		}
		
	}
	//printf("RESA: %d\n", result);
	return result;
}

__device__ void computeColumnSharedMemory(int index, int* tab, int* sharedArray, int actual)
{
	if (threadIdx.x >= height / BOARD_TYPE_LENGTH)
	{
		return;
	}

	int left[3];
	int mid[3];
	int right[3];
	int *board = sharedArray;

	int upperBound = (height / BOARD_TYPE_LENGTH) / blockDim.x;
	for (int dupa = 0; dupa < upperBound; ++dupa) {

		int col = mod(index - (height / BOARD_TYPE_LENGTH), size);

		left[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		left[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		left[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		col = index;

		mid[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		mid[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		mid[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		col = mod(index + (height / BOARD_TYPE_LENGTH), size);

		right[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		right[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		right[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		int result = computeMidCells(left[1], mid[1], right[1]);
		int sum = (left[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (left[1] & 1 ? 1 : 0)
			+ (left[1] & (1 << 1) ? 1 : 0)
			+ (mid[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (mid[1] & (1 << 1) ? 1 : 0)
			+ (right[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (right[1] & 1 ? 1 : 0)
			+ (right[1] & (1 << 1) ? 1 : 0);

		if (mid[1] & 1) {
			if (sum == 2 || sum == 3) {
				result = result | 1;
			}
		}
		else {
			if (sum == 3) {
				result = result | 1;
			}
		}

		sum = (left[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (left[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (left[2] & 1 ? 1 : 0)
			+ (mid[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (mid[2] & 1 ? 1 : 0)
			+ (right[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (right[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (right[2] & 1 ? 1 : 0);

		if (mid[1] & (1 << BOARD_TYPE_LENGTH - 1)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}
		else {
			if (sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}

		tab[index + dupa*blockDim.x + threadIdx.x] = result;
		
	}
}

__global__ void computeNextStepSharedMemory(int* tab_0, int* tab_1, int actual)
{
	int *board = (actual ? tab_0 : tab_1);
	for (int i = 0; i < width; ++i)
	{
		for (int j = 0; j < (height / BOARD_TYPE_LENGTH); ++j)
		{
			if (actual)
			{
				sharedArray[i*(height / BOARD_TYPE_LENGTH) + j] = tab_1[i*(height / BOARD_TYPE_LENGTH) + j];
			}
			else
			{
				sharedArray[i*(height / BOARD_TYPE_LENGTH) + j] = tab_0[i*(height / BOARD_TYPE_LENGTH) + j];
			}
		}
	}
	for (int i = 0; i < width / gridDim.x; ++i) //ASSERT : width % gridDim.x = 0
	{
		computeColumnSharedMemory(i*gridDim.x*(height / BOARD_TYPE_LENGTH)
			+ blockIdx.x*(height / BOARD_TYPE_LENGTH), board, sharedArray, actual);
	}
	__syncthreads();
}

__device__ void computeColumn(int index, int* tab_0, int* tab_1, int actual)
{
	if (threadIdx.x >= height / BOARD_TYPE_LENGTH)
	{
		return;
	}
	
	int left[3];
	int mid[3];
	int right[3];
	
	int *board = (actual ? tab_1 : tab_0);
	
	int upperBound = (height / BOARD_TYPE_LENGTH) / blockDim.x;
	
	for (int dupa = 0; dupa < upperBound; ++dupa) {
		
		int col = mod(index - (height / BOARD_TYPE_LENGTH), size);
		
		left[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		left[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		left[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		col = index;
		
		mid[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		mid[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		mid[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];
		
		col = mod(index + (height / BOARD_TYPE_LENGTH), size);
		
		right[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		right[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		right[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];
		
		int result = computeMidCells(left[1], mid[1], right[1]);
		int sum = (left[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (left[1] & 1 ? 1 : 0)
			+ (left[1] & (1 << 1) ? 1 : 0)
			+ (mid[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (mid[1] & (1 << 1) ? 1 : 0)
			+ (right[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (right[1] & 1 ? 1 : 0)
			+ (right[1] & (1 << 1) ? 1 : 0);

		if (mid[1] & 1) {
			if (sum == 2 || sum == 3) {
				result = result | 1;
			}
		}
		else {
			if (sum == 3) {
				result = result | 1;
			}
		}
		
		sum = (left[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (left[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (left[2] & 1 ? 1 : 0)
			+ (mid[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (mid[2] & 1 ? 1 : 0)
			+ (right[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (right[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (right[2] & 1 ? 1 : 0);

		if (mid[1] & (1 << BOARD_TYPE_LENGTH - 1)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}
		else {
			if (sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}

		if (actual) {
			tab_0[index + dupa*blockDim.x + threadIdx.x] = result;
		}
		else {
			tab_1[index + dupa*blockDim.x + threadIdx.x] = result;
		}
	}
}

__global__ void computeNextStep(int* tab_0, int* tab_1, int actual)
{
	for (int i = 0; i < width / gridDim.x; ++i) //ASSERT : width % gridDim.x = 0
	{
		computeColumn(i*gridDim.x*(height / BOARD_TYPE_LENGTH)
			+ blockIdx.x*(height / BOARD_TYPE_LENGTH), tab_0, tab_1, actual);
	}
	//printf("H: %d, W: %d\n", height, width);
}

void copyArrayToDevice(universe uni, int* tab_0)
{
	for (int i = 0; i < uni.width; ++i)
	{
		hipMemcpy((tab_0)+i*(uni.height / BOARD_TYPE_LENGTH), uni.board[i], (uni.height / BOARD_TYPE_LENGTH) * sizeof(int), hipMemcpyHostToDevice);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(width), &(uni.width), sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(height), &(uni.height), sizeof(int));
	int sizeHost = (uni.width * uni.height) / BOARD_TYPE_LENGTH;
	hipMemcpyToSymbol(HIP_SYMBOL(size), &sizeHost, sizeof(int));
}

void copyArrayToHost(universe* uni, int* tab_0, int* tab_1, int actual)
{
	for (int i = 0; i < uni->width; ++i)
	{
		if (!actual)
			hipMemcpy(uni->board[i], tab_0+i*uni->height / BOARD_TYPE_LENGTH, (uni->height / BOARD_TYPE_LENGTH) * sizeof(int), hipMemcpyDeviceToHost);
		else
			hipMemcpy(uni->board[i], tab_1+i*uni->height / BOARD_TYPE_LENGTH, (uni->height / BOARD_TYPE_LENGTH) * sizeof(int), hipMemcpyDeviceToHost);
	}
}
