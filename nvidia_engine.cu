#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"


__device__ static int width;
__device__ static int height;
__device__ static int size;


//------ Some computational functions ------//

__device__ int mod(int number, int modulus) {
	if (number % modulus >= 0)
		return number % modulus;
	else
		return modulus + (number % modulus);
}

__device__ int computeMidCells(int left, int mid, int right) {
	int result = 0;

	for (int i = 1; i < BOARD_TYPE_LENGTH - 1; ++i) {
		printf("left: %d, mid: %d, right: %d\n", left, mid, right);
		int j = i - 1;
		int k = i + 1;

		int sum = left & (1 << j) ? 1 : 0
			+ left & (1 << i) ? 1 : 0
			+ left & (1 << k) ? 1 : 0
			+ mid & (1 << j) ? 1 : 0
			+ mid & (1 << k) ? 1 : 0
			+ right & (1 << j) ? 1 : 0
			+ right & (1 << i) ? 1 : 0
			+ right & (1 << k) ? 1 : 0;

		if (mid & (1 << i)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << i);
			}
		}
		else {
			if (sum == 3) {
				result = result | (1 << i);
			}
		}
	}

	return result;
}

__device__ void computeColumn(int index, world* w)
{
	printf("index: %d\n", index);
	if (threadIdx.x >= height / BOARD_TYPE_LENGTH)
	{
		printf("In return Index: %d\n", index);
		return;
	}
	//printf("bd: %d\n", blockDim.x);
	//printf("h: %d\n", height);
	//printf("btl: %d\n", BOARD_TYPE_LENGTH);
	//printf("diff: %d\n", (height / BOARD_TYPE_LENGTH) / blockDim.x);

	int left[3];
	int mid[3];
	int right[3];
	printf("1\n");
	int *board = w->actual ? w->tab_1 : w->tab_0;
	printf("2\n");
	int upperBound = (height / BOARD_TYPE_LENGTH) / blockDim.x;
	printf("ub: %d\n", upperBound);
	for (int dupa = 0; dupa < upperBound; ++dupa) {
		printf("DUPA: %d\n", dupa);
		int col = mod(index - (height / BOARD_TYPE_LENGTH), size);
		left[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		left[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		left[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		col = index;
		mid[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		mid[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		mid[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		col = mod(index + (height / BOARD_TYPE_LENGTH), size);
		right[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		right[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		right[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		printf("data gathered index: %d\n", index);

		int result = computeMidCells(left[1], mid[1], right[1]);

		printf("computed mid cells index: %d\n", index);

		int sum = left[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0
			+ left[1] & 1 ? 1 : 0
			+ left[1] & (1 << 1) ? 1 : 0
			+ mid[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0
			+ mid[1] & (1 << 1) ? 1 : 0
			+ right[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0
			+ right[1] & 1 ? 1 : 0
			+ right[1] & (1 << 1) ? 1 : 0;

		if (mid[1] & 1) {
			if (sum == 2 || sum == 3) {
				result = result | 1;
			}
		}
		else {
			if (sum == 3) {
				result = result | 1;
			}
		}
		printf("Summ for index: %d = %d\n", index, sum);
		sum = left[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0
			+ left[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0
			+ left[2] & 1 ? 1 : 0
			+ mid[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0
			+ mid[2] & 1 ? 1 : 0
			+ right[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0
			+ right[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0
			+ right[2] & 1 ? 1 : 0;

		if (mid[1] & (1 << BOARD_TYPE_LENGTH - 1)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}
		else {
			if (sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}

		if (w->actual) {
			w->tab_0[index + dupa*blockDim.x + threadIdx.x] = result;
		}
		else {
			w->tab_1[index + dupa*blockDim.x + threadIdx.x] = result;
		}
		printf("[%d], [%d]: \n", blockIdx.x, threadIdx.x, result);
	}
}

__global__ void computeNextStep(world* w)
{
	for (int i = 0; i < width / gridDim.x; ++i) //ASSERT : width % gridDim.x = 0
	{
		computeColumn(i*gridDim.x*(height / BOARD_TYPE_LENGTH)
			+ blockIdx.x*(height / BOARD_TYPE_LENGTH), w);
	}
	//printf("H: %d, W: %d\n", height, width);
}

int getMooreNeighborhood(int* board_d)
{
	int numNbrs = 0;

	//blockIdx.x * blockDim.x + threadIdx.x;
	//int x = (numer porzadkowy wierzcholka) / (height * BOARD_TYPE_LENGTH);
	//int y = ((numer porzadkowy wierzcholka) - x * (height * BOARD_TYPE_LENGTH));

	return numNbrs;
}

world* copyArrayToDevice(universe uni)
{
	world* w = (world*)malloc(sizeof(world));
	hipMalloc((void**)&(w->tab_0), uni.height*uni.width / sizeof(int));
	hipMalloc((void**)&(w->tab_1), uni.height*uni.width / sizeof(int));
	w->actual = 0;
	for (int i = 0; i < uni.width; ++i)
	{
		hipMemcpy(&(w->tab_0) + i*uni.height / BOARD_TYPE_LENGTH, uni.board[i], uni.height / BOARD_TYPE_LENGTH, hipMemcpyHostToDevice);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(width), &(uni.width), sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(height), &(uni.height), sizeof(int));
	int sizeHost = (uni.width * uni.height) / BOARD_TYPE_LENGTH;
	hipMemcpyToSymbol(HIP_SYMBOL(size), &sizeHost, sizeof(int));

	return w;
}

void copyArrayToHost(world* w, universe* uni)
{
	for (int i = 0; i < uni->width; ++i)
	{
		if (!w->actual)
			hipMemcpy(uni->board[i], &(w->tab_0) + i*uni->height / BOARD_TYPE_LENGTH, uni->height / BOARD_TYPE_LENGTH, hipMemcpyDeviceToHost);
		else
			hipMemcpy(uni->board[i], &(w->tab_1) + i*uni->height / BOARD_TYPE_LENGTH, uni->height / BOARD_TYPE_LENGTH, hipMemcpyDeviceToHost);
	}
}