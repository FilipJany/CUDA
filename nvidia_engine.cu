#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"


__device__ static int width;
__device__ static int height;
__device__ static int size;


//------ Some computational functions ------//

__device__ int mod(int number, int modulus) {
	if (number % modulus >= 0)
		return number % modulus;
	else
		return modulus + (number % modulus);
}

__device__ int computeMidCells(int left, int mid, int right) {
	int result = 0;

	for (int i = 1; i < BOARD_TYPE_LENGTH - 1; ++i) {
		//printf("left: %d, mid: %d, right: %d\n", left, mid, right);
		int j = i - 1;
		int k = i + 1;

		int sum = (left & (1 << j) ? 1 : 0)
			+ (left & (1 << i) ? 1 : 0)
			+ (left & (1 << k) ? 1 : 0)
			+ (mid & (1 << j) ? 1 : 0)
			+ (mid & (1 << k) ? 1 : 0)
			+ (right & (1 << j) ? 1 : 0)
			+ (right & (1 << i) ? 1 : 0)
			+ (right & (1 << k) ? 1 : 0);
		//if (blockIdx.x > 9 && blockIdx.x < 14)
			//printf("%d\t", sum);
		if (mid & (1 << i)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << i);
			}
		}
		else {
			
			if (sum == 3) {
				//printf("aliveCMC: %d\n", mid);
				result = result | (1 << i);
			}
		}
		
	}
	//printf("RESA: %d\n", result);
	return result;
}

__device__ void computeColumn(int index, int* tab_0, int* tab_1, int actual)
{
	//printf("index: %d\n", index);
	//printf("w: %d\n", width);
	//printf("h: %d\n", height);

	if (threadIdx.x >= height / BOARD_TYPE_LENGTH)
	{
		//printf("In return Index: %d\n", index);
		return;
	}
	//printf("bd: %d\n", blockDim.x);
	//printf("h: %d\n", height);
	//printf("btl: %d\n", BOARD_TYPE_LENGTH);
	//printf("diff: %d\n", (height / BOARD_TYPE_LENGTH) / blockDim.x);

	int left[3];
	int mid[3];
	int right[3];
	//printf("ACT: %d\n", actual);
	int *board = (actual ? tab_1 : tab_0);
	//printf("B: %d\n", board);
	//printf("t0: %d\n", tab_0);
	//printf("t1: %d\n", tab_1);
	//printf("2\n");
	int upperBound = (height / BOARD_TYPE_LENGTH) / blockDim.x;
	//printf("ub: %d\n", upperBound);
	for (int dupa = 0; dupa < upperBound; ++dupa) {
		//printf("DUPA: %d\n", dupa);
		int col = mod(index - (height / BOARD_TYPE_LENGTH), size);
		//printf("col left: %d\n", col);

		//printf("L0: %d\n", col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH)));
		left[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		//printf("L1: %d\n", col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH)));
		left[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		//printf("L2: %d\n", col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH)));
		left[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];

		col = index;
		//printf("col mid: %d\n", col);
		mid[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		mid[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		//printf("%d\n", col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH)));
		//if (mid[1] != 0)
		//	printf("m1: %d\n", mid[1]);
		mid[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];
		
		col = mod(index + (height / BOARD_TYPE_LENGTH), size);
		//printf("col right: %d\n", col);
		right[0] = board[col + mod(dupa*blockDim.x + threadIdx.x - 1, (height / BOARD_TYPE_LENGTH))];
		right[1] = board[col + mod(dupa*blockDim.x + threadIdx.x, (height / BOARD_TYPE_LENGTH))];
		right[2] = board[col + mod(dupa*blockDim.x + threadIdx.x + 1, (height / BOARD_TYPE_LENGTH))];
		
		//printf("data gathered index: %d\n", index);

		int result = computeMidCells(left[1], mid[1], right[1]);

		//printf("computed mid cells index: %d\n", index);

		int sum = (left[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (left[1] & 1 ? 1 : 0)
			+ (left[1] & (1 << 1) ? 1 : 0)
			+ (mid[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (mid[1] & (1 << 1) ? 1 : 0)
			+ (right[0] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (right[1] & 1 ? 1 : 0)
			+ (right[1] & (1 << 1) ? 1 : 0);

		if (mid[1] & 1) {
			if (sum == 2 || sum == 3) {
				result = result | 1;
			}
		}
		else {
			if (sum == 3) {
				result = result | 1;
			}
		}
		//printf("Summ for index: %d = %d\n", index, sum);
		sum = (left[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (left[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (left[2] & 1 ? 1 : 0)
			+ (mid[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (mid[2] & 1 ? 1 : 0)
			+ (right[1] & (1 << BOARD_TYPE_LENGTH - 2) ? 1 : 0)
			+ (right[1] & (1 << BOARD_TYPE_LENGTH - 1) ? 1 : 0)
			+ (right[2] & 1 ? 1 : 0);

		if (mid[1] & (1 << BOARD_TYPE_LENGTH - 1)) {
			if (sum == 2 || sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}
		else {
			if (sum == 3) {
				result = result | (1 << BOARD_TYPE_LENGTH - 1);
			}
		}

		if (actual) {
			tab_0[index + dupa*blockDim.x + threadIdx.x] = result;
			//printf("tab0[%d] = %d\n", index + dupa*blockDim.x + threadIdx.x, tab_0[index + dupa*blockDim.x + threadIdx.x]);
		}
		else {
			tab_1[index + dupa*blockDim.x + threadIdx.x] = result;
			//printf("tab1[%d] = %d\n", index + dupa*blockDim.x + threadIdx.x, tab_1[index + dupa*blockDim.x + threadIdx.x]);
		}
		//printf("[%d], [%d]: %d\n", blockIdx.x, threadIdx.x, result);
	}
}

__global__ void computeNextStep(int* tab_0, int* tab_1, int actual)
{
	for (int i = 0; i < width / gridDim.x; ++i) //ASSERT : width % gridDim.x = 0
	{
		computeColumn(i*gridDim.x*(height / BOARD_TYPE_LENGTH)
			+ blockIdx.x*(height / BOARD_TYPE_LENGTH), tab_0, tab_1, actual);
	}
	//printf("H: %d, W: %d\n", height, width);
}

void copyArrayToDevice(universe uni, int* tab_0)
{
	for (int i = 0; i < uni.width; ++i)
	{
		hipMemcpy((tab_0)+i*(uni.height / BOARD_TYPE_LENGTH), uni.board[i], (uni.height / BOARD_TYPE_LENGTH) * sizeof(int), hipMemcpyHostToDevice);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(width), &(uni.width), sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(height), &(uni.height), sizeof(int));
	int sizeHost = (uni.width * uni.height) / BOARD_TYPE_LENGTH;
	hipMemcpyToSymbol(HIP_SYMBOL(size), &sizeHost, sizeof(int));
	//for (int i = 0; i < uni.width; ++i)
	//{
		//printf("ub[%d] - tab[%d]\n", uni.board[i][0], tab_0[i]);
	//}
}

void copyArrayToHost(universe* uni, int* tab_0, int* tab_1, int actual)
{
	for (int i = 0; i < uni->width; ++i)
	{
		if (!actual)
			hipMemcpy(uni->board[i], tab_0+i*uni->height / BOARD_TYPE_LENGTH, (uni->height / BOARD_TYPE_LENGTH) * sizeof(int), hipMemcpyDeviceToHost);
		else
			hipMemcpy(uni->board[i], tab_1+i*uni->height / BOARD_TYPE_LENGTH, (uni->height / BOARD_TYPE_LENGTH) * sizeof(int), hipMemcpyDeviceToHost);
	}
}
