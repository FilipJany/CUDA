#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "gol_engine.h"
#include "nvidia_engine.h"


__device__ static int width;
__device__ static int height;


//------ Some computational functions ------//

int mod(int number, int modulus) {
	return abs(number % modulus);
}

__device__ void computeColumn(int index, world* w)
{

}

__global__ void computeNextStep(world* w)
{
	for (int i = 0; i < width / blockDim.x; ++i)
	{
		computeColumn(i*gridDim.x*blockDim.x*height / BOARD_TYPE_LENGTH
			+ threadIdx.x*height / BOARD_TYPE_LENGTH, w);
	}
	//printf("H: %d, W: %d\n", height, width);
}

int getMooreNeighborhood(int* board_d)
{
    int numNbrs = 0;
	
	//blockIdx.x * blockDim.x + threadIdx.x;
    //int x = (numer porzadkowy wierzcholka) / (height * BOARD_TYPE_LENGTH);
    //int y = ((numer porzadkowy wierzcholka) - x * (height * BOARD_TYPE_LENGTH));

    return numNbrs;
}

world* copyArrayToDevice(universe uni)
{
	world* w = (world*)malloc(sizeof(world));
	
	hipMalloc((void**) &(w->tab_0), uni.height*uni.width / sizeof(int));
	hipMalloc((void**) &(w->tab_1), uni.height*uni.width / sizeof(int));
	w->actual = 0;
	for(int i=0; i < uni.width; ++i)
	{
		hipMemcpy(&(w->tab_0)+i*uni.height/BOARD_TYPE_LENGTH, uni.board[i], uni.height/BOARD_TYPE_LENGTH, hipMemcpyHostToDevice);
	}

	hipMemcpyToSymbol(HIP_SYMBOL(width), &(uni.width),sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(height), &(uni.height), sizeof(int));

	return w;
}

void copyArrayToHost(world* w, universe* uni)
{
	for(int i = 0; i < uni->width; ++i)
	{
		if (!w->actual)
			hipMemcpy(uni->board[i], &(w->tab_0)+i*uni->height/BOARD_TYPE_LENGTH, uni->height/BOARD_TYPE_LENGTH, hipMemcpyDeviceToHost);
		else
			hipMemcpy(uni->board[i], &(w->tab_1) + i*uni->height / BOARD_TYPE_LENGTH, uni->height / BOARD_TYPE_LENGTH, hipMemcpyDeviceToHost);
	}
}
